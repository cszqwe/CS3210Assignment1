/**
 * 
 * Matrix Multiplication - CUDA for GPUs
 *
 * CS3210
 *
 **/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <assert.h>

int size;
#define BLOCK_SIZE 32
typedef struct
{
	float ** element;
} matrix;


long long wall_clock_time()
{
#ifdef __linux__
	struct timespec tp;
	clock_gettime(CLOCK_REALTIME, &tp);
	return (long long)(tp.tv_nsec + (long long)tp.tv_sec * 1000000000ll);
#else
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (long long)(tv.tv_usec * 1000 + (long long)tv.tv_sec * 1000000000ll);
#endif
}

/**
 * Allocates memory for a matrix of size SIZE
 * The memory is allocated row-major order, i.e. 
 *  elements from the same row are allocated at contiguous 
 *  memory addresses.
 **/
void allocate_matrix(matrix* m)
{
	int i;
	hipError_t rc;
	
	// allocate array for all the rows
	rc = hipMallocManaged((void**)&(m->element), sizeof(float*) * size);
	if (rc != hipSuccess)
	{
		fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(rc));
		exit(1);
	}
	
	// allocate an array for each row of the matrix
	for (i = 0; i < size; i++)
	{
		rc = hipMallocManaged((void**)&(m->element[i]), sizeof(float) * size);
		if (rc != hipSuccess)
		{
			fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(rc));
			exit(1);
		}
	}
}

/**
 * Free the memory allocated for a matrix.
 **/
void free_matrix(matrix* m) {
	int i;
	for (i = 0; i < size; i++)
		hipFree(m->element[i]);
	hipFree(m->element);
}

/**
 * Initializes the elements of the matrix with
 * random values between 0 and 9
 **/
void init_matrix(matrix m)
{
	int i, j;
	
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
		{
			m.element[i][j] = rand() % 10;
		}
}

/**
 * Initializes the elements of the matrix with
 * element 0.
 **/
void init_matrix_zero(matrix m)
{
	int i, j;
	
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
		{
			m.element[i][j] = 0.0;
		}
}


/**
 * Multiplies matrix @a with matrix @b storing
 * the result in matrix @result
 * 
 * The multiplication algorithm is the O(n^3) 
 * algorithm
 */
void mm(matrix a, matrix b, matrix result)
{
	int i, j, k;
	
	// Do the multiplication
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
			for(k = 0; k < size; k++)
				result.element[i][j] += a.element[i][k] * b.element[k][j];
}

/**
 * Each kernel computes the result element (i,j).
 */
__global__ void mm_kernel(matrix a, matrix b, matrix result, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x; 
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k;

	if (i >= size || j >= size)
		return;

	for(k = 0; k < size; k++)
		result.element[i][j] += a.element[i][k] * b.element[k][j];
}
__global__ void mm_improved(matrix a, matrix b, matrix result, int size){
	  /*Use shared memory instead of load from global memory each time.
	  The shared memeory would be shared by all the threads inside one block
	  */
        __shared__ float matA[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float matB[BLOCK_SIZE][BLOCK_SIZE];
        const int tidr = threadIdx.x;
        const int tidc = threadIdx.y;
        const int bidr = blockIdx.x * BLOCK_SIZE;
        const int bidc = blockIdx.y * BLOCK_SIZE;
        float tmp = 0;

        for (int i = 0; i < size; i += BLOCK_SIZE){
            matA[tidr][tidc] = a.element[tidr+bidr][tidc+i];
            matB[tidr][tidc] = b.element[tidr+i][tidc + bidc];
            //Each time, all the threads would form two shared matrix, and use the shared matrix to calculate paritial answers.
		__syncthreads();

            for (int j = 0; j < BLOCK_SIZE; j++){
                tmp += matA[tidr][j] * matB[j][tidc];
            }
		//After all threads finish the calculation of these two sub matrix, they would move on to the next step.
            __syncthreads();
        }
        result.element[tidr+bidr][tidc+bidc] = tmp;


}
void print_matrix(matrix m)
{
	int i, j;
	
	for (i = 0; i < size; i++)
	{
		printf("row %4d: ", i);
		for (j = 0; j < size; j++)
			printf("%6.2f  ", m.element[i][j]);
		printf("\n");
	}
}



void work()
{
	matrix a, b, result1, result2, result3;
	long long before, after;
	int correct, i, j, dim;
	hipError_t rc;

	// Allocate memory for matrices
	allocate_matrix(&a);
	allocate_matrix(&b);
	allocate_matrix(&result1);
	allocate_matrix(&result2);	
        allocate_matrix(&result3);
	// Initialize matrix elements
	init_matrix(a);
	init_matrix(b);

	// Perform sequential matrix multiplication
	before = wall_clock_time();
	mm(a, b, result1);
	after = wall_clock_time();
        fprintf(stderr, "Matrix multiplication on CPU took %1.2f seconds\n", ((float)(after - before))/1000000000);

	// Perform CUDA matrix  multiplication
	dim3 block(32, 32);			// a block of 32 x 32 CUDA threads
	dim = (size % 32 == 0) ? size / 32 : size / 32 + 1; 
	dim3 grid(dim, dim);	// a grid of CUDA thread blocks
	before = wall_clock_time();
	mm_improved<<<grid, block>>>(a, b, result2, size);
	hipDeviceSynchronize();
	after = wall_clock_time();
	fprintf(stderr, "Matrix multiplication on GPU took %1.2f seconds\n", ((float)(after - before))/1000000000);
        before = wall_clock_time();
        mm_kernel<<<grid,block>>>(a,b,result3,size);
        hipDeviceSynchronize();
        after = wall_clock_time();
        fprintf(stderr, "Matrix multiplication on GPU took %1.2f seconds\n", ((float)(after - before))/1000000000);
	// was there any error?
        rc = hipGetLastError();
        if (rc != hipSuccess)
                printf("Last CUDA error %s\n", hipGetErrorString(rc));

	// Compare the results
	correct = 1;
	for (i = 0; correct && i < size; i++)
		for (j = 0; j < size; j++)
			if ((result1.element[i][j] != result3.element[i][j])||(result1.element[i][j] != result2.element[i][j])) {
				correct = 0;
				break;
			}

	if (correct)
		printf("The result matrices are identical!\n");
	else
		printf("Difference in result matrices at element (%d, %d)!\n", i, j);

	free_matrix(&a);
	free_matrix(&b);
	free_matrix(&result1);
	free_matrix(&result2);
}


int main(int argc, char ** argv)
{
	srand(0); 

	printf("Usage: %s <size>\n", argv[0]);
    
	if (argc >= 2)
		size = atoi(argv[1]);
	else
		size = 1024;
		
	fprintf(stderr,"Sequential matrix multiplication of size %d\n", size);
    
	// Multiply the matrices
	work();

	return 0;
}
